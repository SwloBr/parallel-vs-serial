#include <iostream>
#include <vector>
#include <cstdlib>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

void createNotes(std::vector<double>& notas, int n) {
    for (int i = 0; i < n; ++i) {
        double nota = (static_cast<double>(rand()) / RAND_MAX) * 10.0;
        notas[i] = std::round(nota * 100.0) / 100.0;
    }
}

__global__ void takeRangeKernel(const double* notas, double min, double max, int n, int* count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        if (notas[idx] > min && notas[idx] <= max) {
            atomicAdd(count, 1);
        }
    }
}

int main(int argc, char* argv[]) {
    const int N = 100000000;
    std::vector<double> notas(N);
    createNotes(notas, N);
    printf("Quantidade de notas: %d\n", N);

    auto start = std::chrono::high_resolution_clock::now();

    double* d_notas;
    hipMalloc(&d_notas, N * sizeof(double));
    hipMemcpy(d_notas, notas.data(), N * sizeof(double), hipMemcpyHostToDevice);

    // Definindo o tamanho dos blocos e grid
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    int* d_count;
    hipMalloc(&d_count, sizeof(int));
    int sumResult = 0;

    for (int i = 0; i < 10; ++i) {
        hipMemset(d_count, 0, sizeof(int));

        if (i == 0) {
            takeRangeKernel<<<numBlocks, blockSize>>>(d_notas, -0.9, i + 1, N, d_count);
        } else {
            takeRangeKernel<<<numBlocks, blockSize>>>(d_notas, i, i + 1, N, d_count);
        }

        // Transferência do resultado após todas as execuções
        int count;
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);
        sumResult += count;

        printf("Notas entre %d.1 e %d: %d\n", i, i + 1, count);
    }

    printf("\nTotal de valores é %d\n", sumResult);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "\nTempo de execução: " << duration.count() << " segundos" << std::endl;

    hipFree(d_count);
    hipFree(d_notas);


    return 0;
}
